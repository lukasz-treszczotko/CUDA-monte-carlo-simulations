

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>


#define PI 3.14159265358979323846

#define N 100000
#define BLOCK_SIZE 1024

__device__ void BoxMuller(float u1, float u2, float *n1, float *n2)
{
	float r = sqrtf(-2*logf(u1));
	float theta = 2*PI*(u2);
	*n1 = r*sinf(theta);
	*n2 = r*cosf(theta);
}

__global__ void norm_transform(float *dev_u1, float *dev_u2, 
		float *dev_n1, float *dev_n2, int size)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	if (tid < size)
	{
		float res1;
		float res2;
		BoxMuller(dev_u1[tid], dev_u2[tid], &res1, &res2);
		dev_n1[tid] = res1;
		dev_n2[tid] = res2;
	}
	
}


int main()
{
	int dev_count;
	hipGetDeviceCount(&dev_count);
	printf("Number of CUDA-capable devices: %d.\n", dev_count);
	hipDeviceProp_t dev_prop;
	for (int j=0; j<dev_count; j++)
	{
		hipGetDeviceProperties(&dev_prop, j);
		printf("Device number %d has max %d threads per block.\n", j, dev_prop.maxThreadsPerBlock);
		printf("Device number %d has %d multiprocessors.\n", j, dev_prop.multiProcessorCount);
	}
	int i;
	
	
	hiprandGenerator_t gen1, gen2;
	float *dev_u1, *dev_u2, *host_u1, *host_u2;
	float *dev_n1, *dev_n2, *host_n1, *host_n2;

	// allocate memory on the host
	//host_u1 = (float*)calloc(N, sizeof(float));
	//host_u2 = (float*)calloc(N, sizeof(float));
	
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	host_n1 = (float*)calloc(N, sizeof(float));
	host_n2 = (float*)calloc(N, sizeof(float));
		

	// allocate memory on the device
	hipMalloc((void**)&dev_u1, N * sizeof(float));
	hipMalloc((void**)&dev_u2, N * sizeof(float));
	
	hipMalloc((void**)&dev_n1, N * sizeof(float));
	hipMalloc((void**)&dev_n2, N * sizeof(float));
	
	
	// create a mersenne twister
	hiprandCreateGenerator(&gen1, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandCreateGenerator(&gen2, HIPRAND_RNG_PSEUDO_MTGP32);

	// set seed
	hiprandGenerateUniform(gen1, dev_u1, N);
	hiprandGenerateUniform(gen2, dev_u2, N);
	
	int numBlocks = ceil(float(N) / BLOCK_SIZE);
	
	// box muller transform
	norm_transform<<<numBlocks, BLOCK_SIZE>>>(dev_u1, dev_u2, dev_n1, dev_n2, N);
	

	// copy device memory to host
	hipMemcpy(host_n1, dev_n1, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(host_n2, dev_n2, N * sizeof(float), hipMemcpyDeviceToHost);
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	printf("Time elapsed to generate 2 x %d normal variables: %f seconds.\n",  N, elapsedTime/1000.0);

	printf("Random normal draws: \n");
	for (i = 0; i < 10; i++)
	{
		printf(" %1.4f  %1.4f\n", host_n1[i], host_n2[i]);
	}
	printf("\n");
	hiprandDestroyGenerator(gen1);
	hiprandDestroyGenerator(gen2);
	hipDeviceReset();
	free(host_n1);
	free(host_n2);
	
	return 0;
}
